#include "hip/hip_runtime.h"
#define K 5
#define intile 8
#define outtile 4

__global__ void convNaive(float *Y, float *X, float *F, int x0, int x1, int f0, int f1){

    // Same convolution

    int row = threadIdx.y + blockIdx.y*blockDim.y;
    int col = threadIdx.x + blockIdx.x*blockDim.x;

    if(row<x0 && col<x1){

        float res = 0.0f;

        for(int i=0; i<f0; i++){
            for(int j=0; j<f1; j++){
                
                int rid = row - f0/2 + i;
                int cid = col - f1/2 + j;

                if(rid>=0 && rid<x0 && cid>=0 && cid<x1){
                    res += X[rid*x1+ cid]*F[i*f1+j];
                }

            }
        }
        Y[row*x1 + col] = res;
    }

}

__global__ convShared(float *Y, float *X, float *F, int x0, int x1, int f0, int f1){
    
    // indices based on output tile
    int row = threadIdx.y + blockIdx.y*outtile;
    int col = threadIdx.x + blockIdx.x*outtile;

    __shared__ float sm[intile*intile];
    int rid = row-k/2;
    int cid = col-k/2;
    // load into shared memory based on intile
    if(rid>=0 && rid<x0 && cid>=0 && cid<x1){
        sm[threadIdx.y*intile + threadIdx.x] = X[rid*x1 + cid];
    }
    else{
        sm[threadIdx.y*intile + threadIdx.x] = 0.0f;
    }
    
    __syncthreads();
    
    //compute conv from sm
    if(threadIdx.x<outtile && threadIdx.y<outtile && row<x0 && col<x1){
                
        float res = 0.0f;
            for(int i=0; i<f0; i++){
                for(int j=0; j<f1; j++){
                    res += F[i*f1+j]*sm[(row+i)*x1 + col+j];
                }
            }
        Y[row*x1+col] = res;
    } 

}